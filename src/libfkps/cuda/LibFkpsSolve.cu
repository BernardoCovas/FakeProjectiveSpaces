#include <LibFkps.h>
#include <libfkpsconfig.h>

#include "../LibFkps.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thread>
#include <vector>


void cudaSolve(FKPS _lib, FKPSBatch _batch)
{
	hipError_t resCode;
	LibFkpsBatch_t* batch = (LibFkpsBatch_t*)_batch;
	LibFkps_t* lib = (LibFkps_t*)_lib;

	hipFunction_t* function = (hipFunction_t*)lib->function;

	int devBatchSize = batch->batchSize;
	hipDeviceptr_t devStateV, devNOutRes, devOutResV;

	hipMalloc(&devStateV, sizeof(int) * batch->N);
	hipMalloc(&devNOutRes, sizeof(int));
	hipMalloc(&devOutResV, sizeof(int) * batch->batchSize);

	hipMemcpyHtoD(devStateV, batch->v, sizeof(int) * batch->N);

	void* args[] = { &devBatchSize, &devStateV, &devNOutRes, &devOutResV };
	hipModuleLaunchKernel(*function, 1, 1, 1, 1, 1, 1, 0, NULL, args, 0);

	hipCtxSynchronize();

	int* hostNOutRes = new int;
	int* hostOutResV;

	hipMemcpyDtoH(&hostNOutRes, devNOutRes, sizeof(int));
	hostOutResV = new int[*hostNOutRes];
	hipMemcpyDtoH(&hostOutResV, devOutResV, sizeof(int));

	hipFree(devStateV);
	hipFree(devNOutRes);
	hipFree(devOutResV);
	
	delete hostNOutRes;
	delete[] hostOutResV;
}


LibFkpsErr_t LibFkpsSolve(FKPS lib, bool verbose)
{
	LibFkpsErr_t errCode;

	errCode = LibFkpsGenerate(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;

	errCode = LibFkpsCompile(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;

	errCode = LibFkpsLoad(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;


	FKPSBatch state;
	std::vector<FKPSBatch> batchV;
	std::vector<std::thread*> threadV;

	errCode = LibFkpsBatchInit(lib, &state);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;
	
	while (true)
	{
		FKPSBatch batch;
		if (LibFkpsBatchNew(state, &batch) == LIBFKPS_ERR_PARTITION_END)
			break;
		cudaSolve(lib, batch);
		batchV.push_back(batch);
	}

	for (FKPSBatch batch : batchV)
		LibFkpsBatchFree(batch);
	
	LibFkpsBatchFree(state);
	return LIBFKPS_ERR_SUCCESS;
}