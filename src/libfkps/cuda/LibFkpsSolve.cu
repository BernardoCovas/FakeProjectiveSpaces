#include <LibFkps.h>
#include <libfkpsconfig.h>

#include "../LibFkps.hh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thread>
#include <vector>


void cudaSolve(FKPS _lib, FKPSBatch _batch)
{
	hipError_t resCode;
	LibFkpsBatch_t* batch = (LibFkpsBatch_t*)_batch;
	LibFkps_t* lib = (LibFkps_t*)_lib;

	hipFunction_t* function = (hipFunction_t*)lib->function;

	hipDeviceptr_t* xV		 = new hipDeviceptr_t[batch->batchSize];
	hipDeviceptr_t* outResV = new hipDeviceptr_t[batch->batchSize];

	size_t cudaParamIntVSize = sizeof(int) * batch->N;
	size_t cudaParamIntSize  = sizeof(int);

	for (int i = 0; i < batch->batchSize; i++)
	{
		LibFkpsBatchIncrement(batch);

		hipDeviceptr_t* devX		= xV + i;
		hipDeviceptr_t* devOutRes = outResV + i;

		hipMalloc(devX, cudaParamIntVSize);
		hipMalloc(devOutRes, cudaParamIntSize);

		hipMemcpyHtoD(*devX, batch->v, cudaParamIntVSize);

		void* args[] = {devX, devOutRes};
		resCode = hipLaunchKernel(reinterpret_cast<const void*>(function), 1, 1, args);
	}

	hipCtxSynchronize();

	int* inX = new int[batch->N];
	for (int i = 0; i < batch->batchSize; i++)
	{
		int outRes;
		hipMemcpyDtoH(&outRes, outResV[i], cudaParamIntSize);

		if (outRes == 0)
		{
			hipMemcpyDtoH(inX, xV[i], cudaParamIntVSize);
			if (LibFkpsBatchAdd(batch, inX) == LIBFKPS_ERR_PARTITION_FULL)
				LibFkpsBatchFlush(batch, _lib);
		}
	}
	delete[] inX;
	LibFkpsBatchFlush(batch, _lib);
}


LibFkpsErr_t LibFkpsSolve(FKPS lib, bool verbose)
{
	LibFkpsErr_t errCode;

	errCode = LibFkpsGenerate(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;

	errCode = LibFkpsCompile(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;

	errCode = LibFkpsLoad(lib);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;


	FKPSBatch state;
	std::vector<FKPSBatch> batchV;

	errCode = LibFkpsBatchInit(lib, &state);
	if (errCode != LIBFKPS_ERR_SUCCESS)
		return errCode;
	
	while (true)
	{
		FKPSBatch batch;
		if (LibFkpsBatchNew(state, &batch) == LIBFKPS_ERR_PARTITION_END)
			break;
		batchV.push_back(batch);
		cudaSolve(lib, batch);
	}

	LibFkpsBatchFree(state);
	return LIBFKPS_ERR_SUCCESS;
}