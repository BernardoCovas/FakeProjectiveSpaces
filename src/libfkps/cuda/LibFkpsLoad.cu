#include <LibFkps.h>

#include "../LibFkps.hh"

#include <dlfcn.h>
#include <hip/hip_runtime.h>

LibFkpsErr_t LibFkpsLoad(FKPS _lib)
{
	hipError_t resCode;
	LibFkps_t* lib = (LibFkps_t*)_lib;

	hipCtx_t cntxt = (hipCtx_t)lib->libHandle;
	cuCtxAttach(&cntxt, 0);

	hipModule_t cuModule;
	resCode = hipModuleLoad(&cuModule, lib->libFileName->c_str());
	if (resCode != hipSuccess)
		return LIBFKPS_ERR_LIB_NOT_FOUND;

	hipFunction_t  function;
	resCode = hipModuleGetFunction(&function, cuModule, "cudaAllSolve");
	if (resCode != hipSuccess)
	{
		hipModuleUnload(cuModule);
		lib->function = nullptr;
		return LIBFKPS_ERR_LIB_NOT_FOUND;
	}

	hipFunction_t* functionP = new hipFunction_t;
	*functionP = function;
	lib->function = (void*)functionP;

	return LIBFKPS_ERR_SUCCESS;
}
