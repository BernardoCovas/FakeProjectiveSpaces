#include <LibFkpsConfig.h>
#include <LibFkps.h>

#include "../LibFkps.hh"

#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

#include <fstream>
#include <string>
#include <sstream>


LibFkpsErr_t LibFkpsCompile(FKPS lib)
{
	LibFkps_t* _lib = (LibFkps_t*)lib;

	std::ifstream fileStream;
	std::string cFileContent;
	hiprtcProgram prog = nullptr;
	FILE* libFile = nullptr;
	char* ptx = nullptr;
	size_t ptxSize;


	libFile = fopen(_lib->libFileName->c_str(), "w");

	if (!libFile)
		goto LIB_FAILED;

	fileStream = std::ifstream(_lib->cFileName->c_str());

	if (!fileStream.is_open()) 
		goto LIB_FAILED;

	cFileContent = std::string((std::istreambuf_iterator<char>(fileStream)),
		(std::istreambuf_iterator<char>()));

	hiprtcCreateProgram(&prog, cFileContent.c_str(), _lib->libFileName->c_str(), 0, nullptr, nullptr);
	if (hiprtcCompileProgram(prog, 0, nullptr) != HIPRTC_SUCCESS)
		goto LIB_FAILED;
	
	hiprtcGetCodeSize(prog, &ptxSize);

	ptx = new char[ptxSize];
	hiprtcGetCode(prog, ptx);
	hiprtcDestroyProgram(&prog);

	fprintf(libFile, ptx);

	fclose(libFile);
	delete[] ptx;
	hiprtcDestroyProgram(&prog);
	fileStream.close();

	return LIBFKPS_ERR_SUCCESS;

LIB_FAILED:

	if (!libFile) return LIBFKPS_ERR_LIB_NOT_FOUND;
	fclose(libFile);
	
	if (fileStream.is_open()) fileStream.close();
	else return LIBFKPS_ERR_SRC_NOT_FOUND;

	if (prog) hiprtcDestroyProgram(&prog);
	if (ptx) delete[] ptx;

	return LIBFKPS_ERR_COMPILE;
}

